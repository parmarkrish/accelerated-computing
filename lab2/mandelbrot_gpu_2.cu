// Optional arguments:
//  -r <img_size>
//  -b <max iterations>
//  -i <implementation: {
//      vector,
//      vector_ilp,
//      vector_multicore,
//      vector_multicore_multithread_single_sm,
//      vector_multicore_multithread_full,
//      vector_multicore_multithread_full_ilp,
//    }>

#include <cstdint>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cassert>

constexpr float window_zoom = 1.0 / 10000.0f;
constexpr float window_x = -0.743643887 - 0.5 * window_zoom;
constexpr float window_y = 0.131825904 - 0.5 * window_zoom;
constexpr uint32_t default_max_iters = 2000;

#define COARSENING_FACTOR 3

__host__ __device__ 
uint32_t ceil_div(uint32_t a, uint32_t b) { return (a + b - 1) / b; }

/// <--- your code here --->

// OPTIONAL: Uncomment this block to include your GPU vector implementation
// from Lab 1 for easy comparison.
//
// (If you do this, you'll need to update your code to use the new constants
// 'window_zoom', 'window_x', and 'window_y'.)

#define HAS_VECTOR_IMPL // <~~ keep this line if you want to benchmark the vector kernel!

////////////////////////////////////////////////////////////////////////////////
// Vector

__global__ void mandelbrot_gpu_vector(
    uint32_t img_size,
    uint32_t max_iters,
    uint32_t *out // pointer to GPU memory
) {
    for (uint64_t i = 0; i < img_size; ++i) {
        for (uint64_t j = 0; j < img_size; j += 32) {
            // Get the plane coordinate X for the image pixel.
            float cx = (float(j + threadIdx.x) / float(img_size)) * window_zoom + window_x;
            float cy = (float(i) / float(img_size)) * window_zoom + window_y;

            // Innermost loop: start the recursion from z = 0.
            float x2 = 0.0f;
            float y2 = 0.0f;
            float w = 0.0f;
            uint32_t iters = 0;
            while (x2 + y2 <= 4.0f && iters < max_iters) {
                float x = x2 - y2 + cx;
                float y = w - (x2 + y2) + cy;
                x2 = x * x;
                y2 = y * y;
                float z = x + y;
                w = z * z;
                ++iters;
            }
            // Write result.
            out[i * img_size + j + threadIdx.x] = iters;
        }
    }
}

void launch_mandelbrot_gpu_vector(
    uint32_t img_size,
    uint32_t max_iters,
    uint32_t *out // pointer to GPU memory
) {
    mandelbrot_gpu_vector<<<1, 32>>>(img_size, max_iters, out);
}

////////////////////////////////////////////////////////////////////////////////
// Vector + ILP


__device__ __forceinline__ 
void get_cond_mask(bool* mask, const float* x2, const float* y2) {
    for (uint32_t i = 0; i < COARSENING_FACTOR; i++) {
        mask[i] = (x2[i] + y2[i] <= 4.0f);
    }
}

__device__ __forceinline__
bool any(bool* mask) {
    for (uint32_t i = 0; i < COARSENING_FACTOR; i++) {
        if (mask[i]) return true;
    }
    return false;
}

__global__ void mandelbrot_gpu_vector_ilp(
    uint32_t img_size,
    uint32_t max_iters,
    uint32_t *out /* pointer to GPU memory */
) {
    float cx[COARSENING_FACTOR];
    float x2[COARSENING_FACTOR];
    float y2[COARSENING_FACTOR];
    float w[COARSENING_FACTOR];
    bool mask[COARSENING_FACTOR];
    uint32_t iters[COARSENING_FACTOR];
    for (uint64_t i = 0; i < img_size; ++i) {
        for (uint64_t j = 0; j < img_size; j += 32 * COARSENING_FACTOR) {
            float cy = (float(i) / float(img_size)) * window_zoom + window_y;

            #pragma unroll
            for (uint32_t k = 0; k < COARSENING_FACTOR; k++) {
                cx[k] = (float(j + threadIdx.x + k*32) / float(img_size)) * window_zoom + window_x;
                x2[k] = 0.0f;
                y2[k] = 0.0f;
                w[k] = 0.0f;
                iters[k] = 0;
            }
            for (uint32_t c = 0; c < max_iters; c++) {
                get_cond_mask(mask, x2, y2);
                if (!any(mask)) break;

                #pragma unroll
                for (uint32_t k = 0; k < COARSENING_FACTOR; k++) {
                    float x = x2[k] - y2[k] + cx[k];
                    float y = w[k] - (x2[k] + y2[k]) + cy;
                    x2[k] = x * x;
                    y2[k] = y * y;
                    float z = x + y;
                    w[k] = z * z;
                    if (mask[k]) ++iters[k];
                }
            }
            for (uint32_t k = 0; k < COARSENING_FACTOR; k++) {
                out[i * img_size + (j + threadIdx.x + k*32)] = iters[k];
            }
        }
    }
}

void launch_mandelbrot_gpu_vector_ilp(
    uint32_t img_size,
    uint32_t max_iters,
    uint32_t *out /* pointer to GPU memory */
) {
    mandelbrot_gpu_vector_ilp<<<1, 32>>>(img_size, max_iters, out);
}

////////////////////////////////////////////////////////////////////////////////
// Vector + Multi-core (Full Machine)

__global__ void mandelbrot_gpu_vector_multicore(
    uint32_t img_size,
    uint32_t max_iters,
    uint32_t *out /* pointer to GPU memory */
) {
    uint32_t row_length = ceil_div(img_size, gridDim.x);
    for (uint64_t r = 0; r < row_length; ++r) {
        int i = r + blockIdx.x * row_length;
        if (i >= img_size) break;

        for (uint64_t j = 0; j < img_size; j += 32 * 4) {
            // Get the plane coordinate X for the image pixel.
            float cx = (float(j + threadIdx.x) / float(img_size)) * window_zoom + window_x;
            float cy = (float(i) / float(img_size)) * window_zoom + window_y;

            // Innermost loop: start the recursion from z = 0.
            float x2 = 0.0f;
            float y2 = 0.0f;
            float w = 0.0f;
            uint32_t iters = 0;
            while (x2 + y2 <= 4.0f && iters < max_iters) {
                float x = x2 - y2 + cx;
                float y = w - (x2 + y2) + cy;
                x2 = x * x;
                y2 = y * y;
                float z = x + y;
                w = z * z;
                ++iters;
            }
            // Write result.
            out[i * img_size + j + threadIdx.x] = iters;
        }
    }
}

void launch_mandelbrot_gpu_vector_multicore(
    uint32_t img_size,
    uint32_t max_iters,
    uint32_t *out /* pointer to GPU memory */
) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    uint32_t num_SMs = deviceProp.multiProcessorCount;

    mandelbrot_gpu_vector_multicore<<<num_SMs, 4 * 32>>>(img_size, max_iters, out);
}

////////////////////////////////////////////////////////////////////////////////
// Vector + Multi-core + Multi-thread-per-core (Single SM)

#define NUM_WARPS 32

__global__ void mandelbrot_gpu_vector_multicore_multithread_single_sm(
    uint32_t img_size,
    uint32_t max_iters,
    uint32_t *out /* pointer to GPU memory */
) {
    uint32_t row_length = ceil_div(img_size, gridDim.x);
    for (uint64_t r = 0; r < row_length; ++r) {
        int i = r + blockIdx.x * row_length;

        for (uint64_t j = 0; j < img_size; j += 32 * NUM_WARPS) {
            // Get the plane coordinate X for the image pixel.
            float cx = (float(j + threadIdx.x) / float(img_size)) * window_zoom + window_x;
            float cy = (float(i) / float(img_size)) * window_zoom + window_y;

            // Innermost loop: start the recursion from z = 0.
            float x2 = 0.0f;
            float y2 = 0.0f;
            float w = 0.0f;
            uint32_t iters = 0;
            while (x2 + y2 <= 4.0f && iters < max_iters) {
                float x = x2 - y2 + cx;
                float y = w - (x2 + y2) + cy;
                x2 = x * x;
                y2 = y * y;
                float z = x + y;
                w = z * z;
                ++iters;
            }
            // Write result.
            if (i < img_size && (j + threadIdx.x) < img_size) {
                out[i * img_size + j + threadIdx.x] = iters;
            }
        }
    }
}


void launch_mandelbrot_gpu_vector_multicore_multithread_single_sm(
    uint32_t img_size,
    uint32_t max_iters,
    uint32_t *out /* pointer to GPU memory */
) {
    mandelbrot_gpu_vector_multicore_multithread_single_sm<<<1, NUM_WARPS * 32>>>(img_size, max_iters, out);
}

////////////////////////////////////////////////////////////////////////////////
// Vector + Multi-core + Multi-thread-per-core (Full Machine)

__global__ void mandelbrot_gpu_vector_multicore_multithread_full(
    uint32_t img_size,
    uint32_t max_iters,
    uint32_t *out /* pointer to GPU memory */
) {
    uint32_t row_length = ceil_div(img_size, gridDim.x);
    for (uint64_t r = 0; r < row_length; ++r) {
        int i = r + blockIdx.x * row_length;

        for (uint64_t j = 0; j < img_size; j += 32 * NUM_WARPS) {
            // Get the plane coordinate X for the image pixel.
            float cx = (float(j + threadIdx.x) / float(img_size)) * window_zoom + window_x;
            float cy = (float(i) / float(img_size)) * window_zoom + window_y;

            // Innermost loop: start the recursion from z = 0.
            float x2 = 0.0f;
            float y2 = 0.0f;
            float w = 0.0f;
            uint32_t iters = 0;
            while (x2 + y2 <= 4.0f && iters < max_iters) {
                float x = x2 - y2 + cx;
                float y = w - (x2 + y2) + cy;
                x2 = x * x;
                y2 = y * y;
                float z = x + y;
                w = z * z;
                ++iters;
            }
            // Write result.
            if (i < img_size && (j + threadIdx.x) < img_size) {
                out[i * img_size + j + threadIdx.x] = iters;
            }
        }
    }
}

void launch_mandelbrot_gpu_vector_multicore_multithread_full(
    uint32_t img_size,
    uint32_t max_iters,
    uint32_t *out /* pointer to GPU memory */
) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    uint32_t num_SMs = deviceProp.multiProcessorCount;

    mandelbrot_gpu_vector_multicore_multithread_full<<<num_SMs, NUM_WARPS * 32>>>(img_size, max_iters, out);
}

////////////////////////////////////////////////////////////////////////////////
// Vector + Multi-core + Multi-thread-per-core + ILP (Full Machine)

__global__ void mandelbrot_gpu_vector_multicore_multithread_full_ilp(
    uint32_t img_size,
    uint32_t max_iters,
    uint32_t *out /* pointer to GPU memory */
) {
    float cy[COARSENING_FACTOR];
    float x2[COARSENING_FACTOR];
    float y2[COARSENING_FACTOR];
    float w[COARSENING_FACTOR];
    bool mask[COARSENING_FACTOR];
    uint32_t iters[COARSENING_FACTOR];

    uint32_t row_length = ceil_div(img_size, gridDim.x);
    assert((row_length % COARSENING_FACTOR == 0));
    for (uint64_t r = 0; r < row_length; r += COARSENING_FACTOR) {
        int i = r + blockIdx.x * row_length;
        for (uint64_t j = 0; j < img_size; j += 32 * NUM_WARPS) {
            float cx = (float(j + threadIdx.x) / float(img_size)) * window_zoom + window_x;

            #pragma unroll
            for (uint32_t k = 0; k < COARSENING_FACTOR; k++) {
                cy[k] = (float(i + k) / float(img_size)) * window_zoom + window_y;
                x2[k] = 0.0f;
                y2[k] = 0.0f;
                w[k] = 0.0f;
                iters[k] = 0;
            }

            for (uint32_t c = 0; c < max_iters; c++) {
                get_cond_mask(mask, x2, y2);
                if (!any(mask)) break;

                #pragma unroll
                for (uint32_t k = 0; k < COARSENING_FACTOR; k++) {
                    float x = x2[k] - y2[k] + cx;
                    float y = w[k] - (x2[k] + y2[k]) + cy[k];
                    x2[k] = x * x;
                    y2[k] = y * y;
                    float z = x + y;
                    w[k] = z * z;
                    if (mask[k]) ++iters[k];
                }
            }
            for (uint32_t k = 0; k < COARSENING_FACTOR; k++) {
                if ((i+k) < img_size && (j + threadIdx.x) < img_size) {
                    out[(i + k) * img_size + (j + threadIdx.x)] = iters[k];
                }
            }
        }
    }
}

void launch_mandelbrot_gpu_vector_multicore_multithread_full_ilp(
    uint32_t img_size,
    uint32_t max_iters,
    uint32_t *out /* pointer to GPU memory */
) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    uint32_t num_SMs = deviceProp.multiProcessorCount;
    
    mandelbrot_gpu_vector_multicore_multithread_full_ilp<<<num_SMs, NUM_WARPS * 32>>>(img_size, max_iters, out);

}

/// <--- /your code here --->

////////////////////////////////////////////////////////////////////////////////
///          YOU DO NOT NEED TO MODIFY THE CODE BELOW HERE.                  ///
////////////////////////////////////////////////////////////////////////////////

#include <algorithm>
#include <chrono>
#include <cstdint>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <sstream>
#include <vector>

// Useful functions and structures.
enum MandelbrotImpl {
    VECTOR,
    VECTOR_ILP,
    VECTOR_MULTICORE,
    VECTOR_MULTICORE_MULTITHREAD_SINGLE_SM,
    VECTOR_MULTICORE_MULTITHREAD_FULL,
    VECTOR_MULTICORE_MULTITHREAD_FULL_ILP,
    ALL
};

// Command-line arguments parser.
int ParseArgsAndMakeSpec(
    int argc,
    char *argv[],
    uint32_t *img_size,
    uint32_t *max_iters,
    MandelbrotImpl *impl) {
    char *implementation_str = nullptr;

    for (int i = 1; i < argc; i++) {
        if (strcmp(argv[i], "-r") == 0) {
            if (i + 1 < argc) {
                *img_size = atoi(argv[++i]);
                if (*img_size % 32 != 0) {
                    std::cerr << "Error: Image width must be a multiple of 32"
                              << std::endl;
                    return 1;
                }
            } else {
                std::cerr << "Error: No value specified for -r" << std::endl;
                return 1;
            }
        } else if (strcmp(argv[i], "-b") == 0) {
            if (i + 1 < argc) {
                *max_iters = atoi(argv[++i]);
            } else {
                std::cerr << "Error: No value specified for -b" << std::endl;
                return 1;
            }
        } else if (strcmp(argv[i], "-i") == 0) {
            if (i + 1 < argc) {
                implementation_str = argv[++i];
                if (strcmp(implementation_str, "vector") == 0) {
                    *impl = VECTOR;
                } else if (strcmp(implementation_str, "vector_ilp") == 0) {
                    *impl = VECTOR_ILP;
                } else if (strcmp(implementation_str, "vector_multicore") == 0) {
                    *impl = VECTOR_MULTICORE;
                } else if (
                    strcmp(
                        implementation_str,
                        "vector_multicore_multithread_single_sm") == 0) {
                    *impl = VECTOR_MULTICORE_MULTITHREAD_SINGLE_SM;
                } else if (
                    strcmp(implementation_str, "vector_multicore_multithread_full") ==
                    0) {
                    *impl = VECTOR_MULTICORE_MULTITHREAD_FULL;
                } else if (
                    strcmp(implementation_str, "vector_multicore_multithread_full_ilp") ==
                    0) {
                    *impl = VECTOR_MULTICORE_MULTITHREAD_FULL_ILP;
                } else if (strcmp(implementation_str, "all") == 0) {
                    *impl = ALL;
                } else {
                    std::cerr << "Error: unknown implementation" << std::endl;
                    return 1;
                }
            } else {
                std::cerr << "Error: No value specified for -i" << std::endl;
                return 1;
            }
        } else {
            std::cerr << "Unknown flag: " << argv[i] << std::endl;
            return 1;
        }
    }
    std::cout << "Testing with image size " << *img_size << "x" << *img_size << " and "
              << *max_iters << " max iterations." << std::endl;

    return 0;
}

// Output image writers: BMP file header structure
#pragma pack(push, 1)
struct BMPHeader {
    uint16_t fileType{0x4D42};   // File type, always "BM"
    uint32_t fileSize{0};        // Size of the file in bytes
    uint16_t reserved1{0};       // Always 0
    uint16_t reserved2{0};       // Always 0
    uint32_t dataOffset{54};     // Start position of pixel data
    uint32_t headerSize{40};     // Size of this header (40 bytes)
    int32_t width{0};            // Image width in pixels
    int32_t height{0};           // Image height in pixels
    uint16_t planes{1};          // Number of color planes
    uint16_t bitsPerPixel{24};   // Bits per pixel (24 for RGB)
    uint32_t compression{0};     // Compression method (0 for uncompressed)
    uint32_t imageSize{0};       // Size of raw bitmap data
    int32_t xPixelsPerMeter{0};  // Horizontal resolution
    int32_t yPixelsPerMeter{0};  // Vertical resolution
    uint32_t colorsUsed{0};      // Number of colors in the color palette
    uint32_t importantColors{0}; // Number of important colors
};
#pragma pack(pop)

void writeBMP(const char *fname, uint32_t img_size, const std::vector<uint8_t> &pixels) {
    uint32_t width = img_size;
    uint32_t height = img_size;

    BMPHeader header;
    header.width = width;
    header.height = height;
    header.imageSize = width * height * 3;
    header.fileSize = header.dataOffset + header.imageSize;

    std::ofstream file(fname, std::ios::binary);
    file.write(reinterpret_cast<const char *>(&header), sizeof(header));
    file.write(reinterpret_cast<const char *>(pixels.data()), pixels.size());
}

std::vector<uint8_t> iters_to_colors(
    uint32_t img_size,
    uint32_t max_iters,
    const std::vector<uint32_t> &iters) {
    uint32_t width = img_size;
    uint32_t height = img_size;
    uint32_t min_iters = max_iters;
    for (uint32_t i = 0; i < img_size; i++) {
        for (uint32_t j = 0; j < img_size; j++) {
            min_iters = std::min(min_iters, iters[i * img_size + j]);
        }
    }
    float log_iters_min = log2f(static_cast<float>(min_iters));
    float log_iters_range =
        log2f(static_cast<float>(max_iters) / static_cast<float>(min_iters));
    auto pixel_data = std::vector<uint8_t>(width * height * 3);
    for (uint32_t i = 0; i < height; i++) {
        for (uint32_t j = 0; j < width; j++) {
            uint32_t iter = iters[i * width + j];

            uint8_t r = 0, g = 0, b = 0;
            if (iter < max_iters) {
                auto log_iter = log2f(static_cast<float>(iter)) - log_iters_min;
                auto intensity = static_cast<uint8_t>(log_iter * 222 / log_iters_range);
                r = 32;
                g = 32 + intensity;
                b = 32;
            }

            auto index = (i * width + j) * 3;
            pixel_data[index] = b;
            pixel_data[index + 1] = g;
            pixel_data[index + 2] = r;
        }
    }
    return pixel_data;
}

// Benchmarking macros and configuration.
#define BENCHPRESS(func, kNumOfOuterIterations, kNumOfInnerIterations, ...) \
    do { \
        std::cout << std::endl << "Running " << #func << " ...\n"; \
        std::vector<double> times(kNumOfOuterIterations); \
        for (size_t i = 0; i < kNumOfOuterIterations; ++i) { \
            auto start = std::chrono::high_resolution_clock::now(); \
            for (size_t j = 0; j < kNumOfInnerIterations; ++j) { \
                func(__VA_ARGS__); \
            } \
            CUDA_CHECK(hipDeviceSynchronize()); \
            auto end = std::chrono::high_resolution_clock::now(); \
            times[i] = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start) \
                           .count() / \
                kNumOfInnerIterations; \
        } \
        std::sort(times.begin(), times.end()); \
        std::stringstream sstream; \
        sstream << std::fixed << std::setw(6) << std::setprecision(2) \
                << times[0] / 1'000'000; \
        std::cout << "  Runtime: " << sstream.str() << " ms" << std::endl; \
    } while (0)

// AUX CUDA check functions.
void cuda_check(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        std::cerr << "CUDA error at " << file << ":" << line << ": "
                  << hipGetErrorString(code) << std::endl;
        exit(1);
    }
}

#define CUDA_CHECK(x) \
    do { \
        cuda_check((x), __FILE__, __LINE__); \
    } while (0)

double difference(
    uint32_t img_size,
    uint32_t max_iters,
    std::vector<uint32_t> &result,
    std::vector<uint32_t> &ref_result) {
    int64_t diff = 0;
    for (uint32_t i = 0; i < img_size; i++) {
        for (uint32_t j = 0; j < img_size; j++) {
            diff +=
                abs(int(result[i * img_size + j]) - int(ref_result[i * img_size + j]));
        }
    }
    return diff / double(img_size * img_size * max_iters);
}

// CPU Scalar Mandelbrot set generation.
// Based on the "optimized escape time algorithm" in
// https://en.wikipedia.org/wiki/Plotting_algorithms_for_the_Mandelbrot_set
void mandelbrot_cpu_scalar(uint32_t img_size, uint32_t max_iters, uint32_t *out) {
    for (uint64_t i = 0; i < img_size; ++i) {
        for (uint64_t j = 0; j < img_size; ++j) {
            float cx = (float(j) / float(img_size)) * window_zoom + window_x;
            float cy = (float(i) / float(img_size)) * window_zoom + window_y;

            float x2 = 0.0f;
            float y2 = 0.0f;
            float w = 0.0f;
            uint32_t iters = 0;
            while (x2 + y2 <= 4.0f && iters < max_iters) {
                float x = x2 - y2 + cx;
                float y = w - (x2 + y2) + cy;
                x2 = x * x;
                y2 = y * y;
                float z = x + y;
                w = z * z;
                ++iters;
            }

            // Write result.
            out[i * img_size + j] = iters;
        }
    }
}

void dump_image(
    const char *fname,
    uint32_t img_size,
    uint32_t max_iters,
    const std::vector<uint32_t> &iters) {
    // Dump result as an image.
    auto pixel_data = iters_to_colors(img_size, max_iters, iters);
    writeBMP(fname, img_size, pixel_data);
}

// Main function.
// Compile with:
// nvcc -O3 -o mandelbrot mandelbrot_gpu_2.cu
int main(int argc, char *argv[]) {
    // Get Mandelbrot spec.
    uint32_t img_size = 1024;
    uint32_t max_iters = default_max_iters;
    enum MandelbrotImpl impl = ALL;
    if (ParseArgsAndMakeSpec(argc, argv, &img_size, &max_iters, &impl))
        return -1;

    // Allocate memory.
    std::vector<uint32_t> ref_result(img_size * img_size);
    std::vector<uint32_t> result_host(img_size * img_size);

    // Compute the reference solution
    mandelbrot_cpu_scalar(img_size, max_iters, ref_result.data());

    // Allocate CUDA memory.
    uint32_t *result_device;
    CUDA_CHECK(hipMalloc(&result_device, img_size * img_size * sizeof(uint32_t)));

    // Test the desired kernels.
#ifdef HAS_VECTOR_IMPL
    if (impl == VECTOR || impl == ALL) {
        CUDA_CHECK(hipMemset(result_device, 0, img_size * img_size * sizeof(uint32_t)));
        BENCHPRESS(
            launch_mandelbrot_gpu_vector,
            1,
            4,
            img_size,
            max_iters,
            result_device);
        // Copy result back.
        CUDA_CHECK(hipMemcpy(
            result_host.data(),
            result_device,
            img_size * img_size * sizeof(uint32_t),
            hipMemcpyDeviceToHost));
        dump_image("out/mandelbrot_gpu_vector.bmp", img_size, max_iters, result_host);
        // Check for correctness.
        std::cout << "  Correctness: average output difference from reference "
                  << difference(img_size, max_iters, result_host, ref_result)
                  << std::endl;
    }
#endif

    if (impl == VECTOR_ILP || impl == ALL) {
        CUDA_CHECK(hipMemset(result_device, 0, img_size * img_size * sizeof(uint32_t)));
        BENCHPRESS(
            launch_mandelbrot_gpu_vector_ilp,
            1,
            2,
            img_size,
            max_iters,
            result_device);
        // Copy result back.
        CUDA_CHECK(hipMemcpy(
            result_host.data(),
            result_device,
            img_size * img_size * sizeof(uint32_t),
            hipMemcpyDeviceToHost));
        dump_image("out/mandelbrot_gpu_vector_ilp.bmp", img_size, max_iters, result_host);
        // Check for correctness.
        std::cout << "  Correctness: average output difference from reference "
                  << difference(img_size, max_iters, result_host, ref_result)
                  << std::endl;
    }


    if (impl == VECTOR_MULTICORE || impl == ALL) {
        CUDA_CHECK(hipMemset(result_device, 0, img_size * img_size * sizeof(uint32_t)));
        BENCHPRESS(
            launch_mandelbrot_gpu_vector_multicore,
            3,
            5,
            img_size,
            max_iters,
            result_device);
        // Copy result back.
        CUDA_CHECK(hipMemcpy(
            result_host.data(),
            result_device,
            img_size * img_size * sizeof(uint32_t),
            hipMemcpyDeviceToHost));
        dump_image(
            "out/mandelbrot_gpu_vector_multicore.bmp",
            img_size,
            max_iters,
            result_host);
        // Check for correctness.
        std::cout << "  Correctness: average output difference from reference "
                  << difference(img_size, max_iters, result_host, ref_result)
                  << std::endl;
    }

    if (impl == VECTOR_MULTICORE_MULTITHREAD_SINGLE_SM || impl == ALL) {
        CUDA_CHECK(hipMemset(result_device, 0, img_size * img_size * sizeof(uint32_t)));
        BENCHPRESS(
            launch_mandelbrot_gpu_vector_multicore_multithread_single_sm,
            3,
            5,
            img_size,
            max_iters,
            result_device);
        // Copy result back.
        CUDA_CHECK(hipMemcpy(
            result_host.data(),
            result_device,
            img_size * img_size * sizeof(uint32_t),
            hipMemcpyDeviceToHost));
        dump_image(
            "out/mandelbrot_gpu_vector_multicore_multithread_single_sm.bmp",
            img_size,
            max_iters,
            result_host);
        // Check for correctness.
        std::cout << "  Correctness: average output difference from reference "
                  << difference(img_size, max_iters, result_host, ref_result)
                  << std::endl;
    }

    if (impl == VECTOR_MULTICORE_MULTITHREAD_FULL || impl == ALL) {
        CUDA_CHECK(hipMemset(result_device, 0, img_size * img_size * sizeof(uint32_t)));
        BENCHPRESS(
            launch_mandelbrot_gpu_vector_multicore_multithread_full,
            3,
            5,
            img_size,
            max_iters,
            result_device);
        // Copy result back.
        CUDA_CHECK(hipMemcpy(
            result_host.data(),
            result_device,
            img_size * img_size * sizeof(uint32_t),
            hipMemcpyDeviceToHost));
        dump_image(
            "out/mandelbrot_gpu_vector_multicore_multithread_full.bmp",
            img_size,
            max_iters,
            result_host);
        // Check for correctness.
        std::cout << "  Correctness: average output difference from reference "
                  << difference(img_size, max_iters, result_host, ref_result)
                  << std::endl;
    }

    if (impl == VECTOR_MULTICORE_MULTITHREAD_FULL_ILP || impl == ALL) {
        CUDA_CHECK(hipMemset(result_device, 0, img_size * img_size * sizeof(uint32_t)));
        BENCHPRESS(
            launch_mandelbrot_gpu_vector_multicore_multithread_full_ilp,
            3,
            5,
            img_size,
            max_iters,
            result_device);
        // Copy result back.
        CUDA_CHECK(hipMemcpy(
            result_host.data(),
            result_device,
            img_size * img_size * sizeof(uint32_t),
            hipMemcpyDeviceToHost));
        dump_image(
            "out/mandelbrot_gpu_vector_multicore_multithread_full_ilp.bmp",
            img_size,
            max_iters,
            result_host);
        // Check for correctness.
        std::cout << "  Correctness: average output difference from reference "
                  << difference(img_size, max_iters, result_host, ref_result)
                  << std::endl;
    }

    // Free CUDA memory.
    CUDA_CHECK(hipFree(result_device));

    return 0;
}