// Tested on RTX A4000
// nvcc -O3 --use_fast_math -std=c++17 -gencode arch=compute_86,code=sm_86 -o matmul matmul_3.cu
#include <chrono>
#include <cstdint>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <map>
#include <random>
#include <sstream>
#include <tuple>
#include <utility>
#include <vector>

void cuda_check(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        std::cerr << "CUDA error at " << file << ":" << line << ": "
                  << hipGetErrorString(code) << std::endl;
        exit(1);
    }
}

#define CUDA_CHECK(x) \
    do { \
        cuda_check((x), __FILE__, __LINE__); \
    } while (0)

__device__ __forceinline__ void cp_async4(void *smem_ptr, const void *glob_ptr) {
    const int BYTES = 16;
    uint32_t smem = static_cast<uint32_t>(__cvta_generic_to_shared(smem_ptr));
    asm volatile(
        "cp.async.cg.shared.global [%0], [%1], %2;" ::"r"(smem),
        "l"(glob_ptr),
        "n"(BYTES));
}

__device__ __forceinline__ void async_memcpy_waitall() {
    asm volatile("cp.async.wait_all;\n" ::);
}

////////////////////////////////////////////////////////////////////////////////
// CPU Reference Implementation (Too slow to actually run!)
//
// void matmul_cpu_naive(
//     int32_t size_i,
//     int32_t size_j,
//     int32_t size_k,
//     float const *a,
//     float const *b,
//     float *c) {
//     for (int32_t i = 0; i < size_i; ++i) {
//         for (int32_t j = 0; j < size_j; ++j) {
//             float sum = 0.0;
//             for (int32_t k = 0; k < size_k; ++k) {
//                 sum += a[i * size_k + k] * b[k * size_j + j];
//             }
//             c[i * size_j + j] = sum;
//         }
//     }
// }

/// <--- your code here --->

/*
    // OPTIONAL: Uncomment this block to include your kernel implementation
    // from Lab 5 for easy comparison.

    ////////////////////////////////////////////////////////////////////////////////
    // Optimized GPU Implementation with Reduction along k (Baseline from Lab 5)

    #define HAS_LAB_5_BASELINE_IMPL // <~~ keep this line if you want to benchmark your Lab 5 kernel!

    namespace matmul_improved_reduce {

    // TODO: your GPU kernels here...

    size_t get_workspace_size(int32_t size_i, int32_t size_j, int32_t size_k) {
        // TODO: your CPU code here
        return 0;
    }

    void launch_matmul_improved_reduce(
        int32_t size_i,
        int32_t size_j,
        int32_t size_k,
        float const *a, // pointer to GPU memory
        float const *b, // pointer to GPU memory
        float *c,       // pointer to GPU memory
        void *workspace // pointer to GPU memory
    ) {
        // TODO: your CPU code here
    }

    } // namespace matmul_improved_reduce
*/

////////////////////////////////////////////////////////////////////////////////
// Tensor Core GPU Implementation

__device__ __forceinline__ void swap(float* &a, float* &b) {
    float* temp = a;
    a = b;
    b = temp;
}

namespace matmul_tensor {

constexpr int REDUCE_DIM = 32;
// constexpr int K_SPLIT_SIZE = 768;  // multiple of REDUCE_SIZE
constexpr int K_SPLIT_SIZE = 1536;  // multiple of REDUCE_SIZE

// must be multiple of 4 to avoid misaligned shared memory write from global memory
constexpr int A_PAD = 4;
constexpr int B_PAD = 8;

// with A_PAD =4 we hit all the banks when loading in a
// a0 banks
// 0  1  2  3
// 4  5  6  7
// 8  9  10 11
// 12 13 14 16 
// 16 17 18 19
// 20 21 22 23
// 24 25 26 27
// 28 29 30 31

// with PAD = 8 we hit every bank when loading b
// b0 banks
// 0  1  2  3  4  5  6  7
// 8  9  10 11 12 13 14 15
// 16 17 18 19 20 21 22 23
// 24 25 26 27 28 29 30 31

__device__ __forceinline__ void load_tiles_to_shared_fp4(
    float* a_shared, 
    float* b_shared, 
    const float* a, 
    const float* b, 
    int size_i,
    int size_j,
    int size_k,
    int tile_idx,
    dim3 tileDim,
    int threadIdx_lin) {

    const int threadIdx_lin4 = 4 * threadIdx_lin;

    const int y_a = tileDim.y*blockIdx.y + (threadIdx_lin4 / REDUCE_DIM);
    const int x_a = tile_idx*REDUCE_DIM + (threadIdx_lin4 % REDUCE_DIM);

    const int y_b = tile_idx*REDUCE_DIM + (threadIdx_lin4 / tileDim.x);
    const int x_b = tileDim.x*blockIdx.x + (threadIdx_lin4 % tileDim.x);

    // void* a_shmem_addr = a_shared + threadIdx_lin4;
    void* a_shmem_addr = a_shared + (REDUCE_DIM + A_PAD) * (threadIdx_lin4 / REDUCE_DIM) + (threadIdx_lin4 % REDUCE_DIM);
    // adding in padding
    void* b_shmem_addr = b_shared + (tileDim.x + B_PAD) * (threadIdx_lin4 / tileDim.x) + (threadIdx_lin4 % tileDim.x);

    if (y_a < size_i && x_a < size_k) {
        const void* a_gmem_addr = a + ((y_a)*size_k + (x_a));
        cp_async4(a_shmem_addr, a_gmem_addr);
    } else {    
        reinterpret_cast<float4*>(a_shmem_addr)[0] = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    }

    if (y_b < size_k && x_b < size_j) {
        const void* b_gmem_addr = b + ((y_b)*size_j + (x_b));
        cp_async4(b_shmem_addr, b_gmem_addr);
    } else {
        reinterpret_cast<float4*>(b_shmem_addr)[0] = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    }
}

__device__ __forceinline__ void mma_16x8x8_4wide(float const *a, float const *b, int* d0, int* d1, int* d2, int* d3, dim3 tileDim, int threadIdx_lin_local) {
    const int a0 = (threadIdx_lin_local % 4) + (REDUCE_DIM + A_PAD) * (threadIdx_lin_local / 4);
    const int a1 = a0 + 8 * (REDUCE_DIM + A_PAD);
    const int a2 = a0 + 4;
    const int a3 = a1 + 4;

    int b0 = (threadIdx_lin_local % 4) * (tileDim.x + B_PAD) + (threadIdx_lin_local / 4);
    int b1 = b0 + 4 * (tileDim.x + B_PAD);

    #pragma unroll
    for (int i = 0; i < 4; i++) {
        asm(
            "mma.sync.aligned.m16n8k8.row.col.f32.tf32.tf32.f32 \
            {%0, %1, %2, %3},     /* 'D' matrix */ \
            {%4, %5, %6, %7},     /* 'A' matrix */ \
            {%8, %9},             /* 'B' matrix */ \
            {%0, %1, %2, %3}      /* 'C' matrix - Same as D  */;"
            : "+r"(d0[i]), "+r"(d1[i]), "+r"(d2[i]), "+r"(d3[i])
            : "r"(__float_as_uint(a[a0])), "r"(__float_as_uint(a[a1])), "r"(__float_as_uint(a[a2])), "r"(__float_as_uint(a[a3])),
            "r"(__float_as_uint(b[b0 + i*8])), "r"(__float_as_uint(b[b1 + i*8]))
        );
    }
}

__device__ __forceinline__ void reduce(
    const float* a_shmem, 
    const float* b_shmem, 
    int* d0, 
    int* d1, 
    int* d2, 
    int* d3, 
    int x_idx_start,
    int y_idx_start,
    dim3 tileDim, 
    int threadIdx_lin_local) {

    const float* a_start = a_shmem + (y_idx_start * (REDUCE_DIM + A_PAD));
    const float* b_start = b_shmem + x_idx_start;

    const int num_tiles = REDUCE_DIM / 8;

    for (int tile_idx = 0; tile_idx < num_tiles; tile_idx++) {
        mma_16x8x8_4wide(a_start, b_start, d0, d1, d2, d3, tileDim, threadIdx_lin_local);
        a_start += 8;
        b_start += 8 * (tileDim.x + B_PAD);
    }
}


__global__ void matmul_tensor(
    int32_t size_i,
    int32_t size_j,
    int32_t size_k,
    float const *a, /* pointer to GPU memory */
    float const *b, /* pointer to GPU memory */
    float *workspace /* pointer to GPU memory */) {

    const dim3 tileDim{4 * blockDim.x, 4 * blockDim.y};
    float* c = workspace + blockIdx.z*size_i*size_j;


    // shared memory setup
    extern __shared__ float shmem[];
    // const int tile_size = tileDim.x * REDUCE_DIM;
    const int a_tile_size = tileDim.y * (REDUCE_DIM + A_PAD);
    const int b_tile_size = REDUCE_DIM * (tileDim.x + B_PAD);
    float* a_shmem0 = shmem;
    float* b_shmem0 = &shmem[a_tile_size];

    float* a_shmem1 = &shmem[a_tile_size + b_tile_size];
    float* b_shmem1 = &shmem[2 * a_tile_size + b_tile_size];

    // warp idxs setup
    const int threadIdx_lin = (threadIdx.y * blockDim.x) + threadIdx.x; // linearize
    const int threadIdx_lin_local = threadIdx_lin % 32;
    const int warp_idx = threadIdx_lin / 32;
    const int y_idx_start = 16 * (warp_idx / 4); 
    const int x_idx_start = 32 * (warp_idx % 4);

    // partial sums
    int d0[4] = {0};
    int d1[4] = {0};
    int d2[4] = {0};
    int d3[4] = {0};

    const uint32_t tile_iters = K_SPLIT_SIZE / REDUCE_DIM;
    const int start_tile_iter = (blockIdx.z) * K_SPLIT_SIZE / REDUCE_DIM;

    // load inital tiles
    load_tiles_to_shared_fp4(a_shmem0, b_shmem0, a, b, size_i, size_j, size_k, start_tile_iter, tileDim, threadIdx_lin);
    async_memcpy_waitall();
    __syncthreads();

    for (int tile_idx = 1; tile_idx < tile_iters; tile_idx++) {
        // load in next tile
        load_tiles_to_shared_fp4(a_shmem1, b_shmem1, a, b, size_i, size_j, size_k, start_tile_iter + tile_idx, tileDim, threadIdx_lin);

        // reduce(t, tileDim, a_shared0, b_shared0, sum);
        reduce(a_shmem0, b_shmem0, d0, d1, d2, d3, x_idx_start, y_idx_start, tileDim, threadIdx_lin_local);

        swap(a_shmem0, a_shmem1);
        swap(b_shmem0, b_shmem1);

        async_memcpy_waitall();
        __syncthreads();
    }

    // last last tile
    reduce(a_shmem0, b_shmem0, d0, d1, d2, d3, x_idx_start, y_idx_start, tileDim, threadIdx_lin_local);
    
    // store sum
    const int y_c = blockIdx.y * tileDim.y + y_idx_start;
    const int x_c = blockIdx.x * tileDim.x + x_idx_start;

    if (y_c >= size_i || x_c >= size_j) return;

    float* c_start = c + (y_c * size_j) + x_c;
    const int c0 = 2 * (threadIdx_lin_local % 4) + size_j * (threadIdx_lin_local / 4);
    const int c1 = c0 + 1;
    const int c2 = c0 + 8 * size_j;
    const int c3 = c1 + 8 * size_j;

    #pragma unroll
    for (int i = 0; i < 4; i++) {
        c_start[c0 + i*8] = __uint_as_float(d0[i]);
        c_start[c1 + i*8] = __uint_as_float(d1[i]);
        c_start[c2 + i*8] = __uint_as_float(d2[i]);
        c_start[c3 + i*8] = __uint_as_float(d3[i]);
    }
}

__global__ void reduce_k(int32_t size_i, int32_t size_j, float* workspace, float* c) {
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (y < size_i && x < size_j) {
        float sum = 0;
        for (int i = 0; i < gridDim.z; i++) {
            sum += (workspace + i*size_i*size_j)[(y)*size_j + (x)];
        }
        c[(y)*size_j + (x)] = sum;
    }
}

size_t get_workspace_size(int32_t size_i, int32_t size_j, int32_t size_k) {
    /* TODO: your CPU code here */
    const int num_splits = (size_k + K_SPLIT_SIZE - 1) / K_SPLIT_SIZE;
    return size_i * size_j * num_splits * 4;
}

void launch_matmul_tensor(
    int32_t size_i,
    int32_t size_j,
    int32_t size_k,
    float const *a, /* pointer to GPU memory */
    float const *b, /* pointer to GPU memory */
    float *c,       /* pointer to GPU memory */
    void *workspace /* pointer to GPU memory */
) {
    /* TODO: your CPU code here */
    const int num_splits = (size_k + K_SPLIT_SIZE - 1) / K_SPLIT_SIZE;
    dim3 block_size(32, 32);
    dim3 tile_size(4 * block_size.x, 4 * block_size.y); // does block_size * k work?
    dim3 num_blocks(
        (size_j + tile_size.x - 1) / tile_size.x,
        (size_i + tile_size.y - 1) / tile_size.y,
        num_splits
    );
    
    // uint32_t shmem_bytes = 2 * 2 * tile_size.x * REDUCE_DIM * sizeof(float);
    uint32_t shmem_bytes = 2 * ((tile_size.y * (REDUCE_DIM + A_PAD)) + (REDUCE_DIM * (tile_size.x + B_PAD))) * sizeof(float);

    CUDA_CHECK(hipFuncSetAttribute(
        reinterpret_cast<const void*>(matmul_tensor),
        hipFuncAttributeMaxDynamicSharedMemorySize, 
        shmem_bytes));
    
    matmul_tensor<<<num_blocks, block_size, shmem_bytes>>>(size_i, size_j, size_k, a, b, (float*) workspace);

    CUDA_CHECK(hipGetLastError());

    dim3 num_blocks_reduce_k(
        (size_j + block_size.x - 1) / block_size.x,
        (size_i + block_size.y - 1) / block_size.y,
        num_splits
    );

    reduce_k<<<num_blocks_reduce_k, block_size>>>(size_i, size_j, (float*) workspace, c);
    CUDA_CHECK(hipGetLastError());
}

}; // namespace matmul_tensor

/// <--- /your code here --->

////////////////////////////////////////////////////////////////////////////////
///          YOU DO NOT NEED TO MODIFY THE CODE BELOW HERE.                  ///
////////////////////////////////////////////////////////////////////////////////

std::vector<float> read_data(std::string const &path, int32_t size) {
    std::ifstream file(path, std::ios::binary);
    std::vector<float> data(size);
    file.read(reinterpret_cast<char *>(data.data()), data.size() * sizeof(float));
    if (file.fail()) {
        std::cerr << "Failed to read " << path << std::endl;
        std::abort();
    }
    return data;
}

template <typename Reset, typename F>
double
benchmark_ms(double target_time_ms, int32_t num_iters_inner, Reset &&reset, F &&f) {
    double best_time_ms = std::numeric_limits<double>::infinity();
    double elapsed_ms = 0.0;
    while (elapsed_ms < target_time_ms) {
        reset();
        CUDA_CHECK(hipDeviceSynchronize());
        auto start = std::chrono::high_resolution_clock::now();
        for (int32_t i = 0; i < num_iters_inner; ++i) {
            f();
        }
        CUDA_CHECK(hipDeviceSynchronize());
        auto end = std::chrono::high_resolution_clock::now();
        double this_ms = std::chrono::duration<double, std::milli>(end - start).count();
        elapsed_ms += this_ms;
        best_time_ms = std::min(best_time_ms, this_ms / num_iters_inner);
    }
    return best_time_ms;
}

struct BenchmarkConfig {
    int32_t size_i;
    int32_t size_j;
    int32_t size_k;
};

struct TestData {
    std::map<std::tuple<int32_t, int32_t>, std::vector<float>> a;
    std::map<std::tuple<int32_t, int32_t>, std::vector<float>> b;
    std::map<std::tuple<int32_t, int32_t, int32_t>, std::vector<float>> c;
};

TestData read_test_data(
    std::string const &test_data_dir,
    std::vector<BenchmarkConfig> const &configs) {
    auto data = TestData{};
    for (auto const &config : configs) {
        auto size_i = config.size_i;
        auto size_j = config.size_j;
        auto size_k = config.size_k;

        auto path_prefix = test_data_dir + "/test_";

        if (data.a.find({size_i, size_k}) == data.a.end()) {
            data.a[{size_i, size_k}] = read_data(
                path_prefix + "a_" + std::to_string(size_i) + "x" +
                    std::to_string(size_k) + ".bin",
                size_i * size_k);
        }

        if (data.b.find({size_k, size_j}) == data.b.end()) {
            data.b[{size_k, size_j}] = read_data(
                path_prefix + "b_" + std::to_string(size_k) + "x" +
                    std::to_string(size_j) + ".bin",
                size_k * size_j);
        }

        if (data.c.find({size_i, size_j, size_k}) == data.c.end()) {
            data.c[{size_i, size_j, size_k}] = read_data(
                path_prefix + "c_" + std::to_string(size_i) + "x" +
                    std::to_string(size_j) + "x" + std::to_string(size_k) + ".bin",
                size_i * size_j);
        }
    }
    return data;
}

struct BenchmarkResults {
    char const *name;
    std::map<std::tuple<int32_t, int32_t, int32_t>, double> elapsed_ms;
};

enum class Phase {
    WARMUP,
    BENCHMARK,
};

template <typename Impl>
void run_config(
    Phase phase,
    TestData const &data,
    BenchmarkConfig const &config,
    BenchmarkResults &results) {
    auto size_i = config.size_i;
    auto size_j = config.size_j;
    auto size_k = config.size_k;

    auto const &a = data.a.at({size_i, size_k});
    auto const &b = data.b.at({size_k, size_j});
    auto const &c = data.c.at({size_i, size_j, size_k});

    float *a_gpu;
    float *b_gpu;
    float *c_gpu;
    CUDA_CHECK(hipMalloc(&a_gpu, size_i * size_k * sizeof(float)));
    CUDA_CHECK(hipMalloc(&b_gpu, size_k * size_j * sizeof(float)));
    CUDA_CHECK(hipMalloc(&c_gpu, size_i * size_j * sizeof(float)));

    CUDA_CHECK(hipMemcpy(
        a_gpu,
        a.data(),
        size_i * size_k * sizeof(float),
        hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(
        b_gpu,
        b.data(),
        size_k * size_j * sizeof(float),
        hipMemcpyHostToDevice));

    size_t workspace_size = Impl::get_workspace_size(size_i, size_j, size_k);
    void *workspace_gpu = nullptr;
    if (workspace_size > 0) {
        CUDA_CHECK(hipMalloc(&workspace_gpu, workspace_size));
        CUDA_CHECK(hipMemset(workspace_gpu, 0, workspace_size));
    }

    if (phase == Phase::BENCHMARK) {
        printf("  %6d  %6d  %6d", size_i, size_j, size_k);
    } else {
        printf("  warmup %6d  %6d  %6d", size_i, size_j, size_k);
    }

    Impl::run(size_i, size_j, size_k, a_gpu, b_gpu, c_gpu, workspace_gpu);

    std::vector<float> c_out_host(size_i * size_j);
    CUDA_CHECK(hipMemcpy(
        c_out_host.data(),
        c_gpu,
        size_i * size_j * sizeof(float),
        hipMemcpyDeviceToHost));

    double mse = 0.0;
    double ref_mean_square = 0.0;
    for (int32_t i = 0; i < size_i; ++i) {
        for (int32_t j = 0; j < size_j; ++j) {
            float diff = c_out_host[i * size_j + j] - c[i * size_j + j];
            mse += diff * diff;
            ref_mean_square += c[i * size_j + j] * c[i * size_j + j];
        }
    }
    mse /= size_i * size_j;
    ref_mean_square /= size_i * size_j;
    float rmse = std::sqrt(mse);
    float rel_rmse = rmse / std::sqrt(ref_mean_square);

    if (phase == Phase::BENCHMARK) {
        printf("  %8.02e", rel_rmse);
    }

    if (rel_rmse > 1e-3) {
        if (phase == Phase::BENCHMARK) {
            printf("  %9s  %7s", "-", "-");
        }
    } else {
        double target_time_ms = 200.0;
        double elapsed_ms = benchmark_ms(
            target_time_ms,
            4,
            [&]() {
                if (workspace_size > 0) {
                    CUDA_CHECK(hipMemset(workspace_gpu, 0, workspace_size));
                }
            },
            [&]() {
                Impl::run(size_i, size_j, size_k, a_gpu, b_gpu, c_gpu, workspace_gpu);
            });

        if (phase == Phase::BENCHMARK) {
            double tflop = 2.0 * size_i * size_k * size_j * 1e-12;
            printf("  %9.02f  %7.02f", elapsed_ms, tflop / (elapsed_ms * 1e-3));

            results.elapsed_ms[{size_i, size_j, size_k}] = elapsed_ms;
        }
    }

    printf("\n");

    CUDA_CHECK(hipFree(a_gpu));
    CUDA_CHECK(hipFree(b_gpu));
    CUDA_CHECK(hipFree(c_gpu));
    if (workspace_size > 0) {
        CUDA_CHECK(hipFree(workspace_gpu));
    }
}

template <typename Impl>
BenchmarkResults run_all_configs(
    Phase phase,
    TestData const &data,
    std::vector<BenchmarkConfig> const &configs) {
    auto results = BenchmarkResults{Impl::name};
    if (phase == Phase::WARMUP) {
        printf("warmup %s:\n\n", Impl::name);
    } else {
        printf("%s:\n\n", Impl::name);
        printf(
            "  %-6s  %-6s  %-6s  %-8s  %-9s  %-7s\n",
            "size_i",
            "size_j",
            "size_k",
            "RRMSE",
            "time (ms)",
            "TFLOP/s");
        printf(
            "  %-6s  %-6s  %-6s  %-8s  %-9s  %-7s\n",
            "------",
            "------",
            "------",
            "--------",
            "---------",
            "-------");
    }
    for (auto const &config : configs) {
        run_config<Impl>(phase, data, config, results);
    }
    printf("\n");
    return results;
}

#ifdef HAS_LAB_5_BASELINE_IMPL

struct MatmulImprovedReduce {
    constexpr static char const *name = "matmul_improved_reduce";

    static size_t get_workspace_size(int32_t size_i, int32_t size_j, int32_t size_k) {
        return matmul_improved_reduce::get_workspace_size(size_i, size_j, size_k);
    }

    static void
    run(int32_t size_i,
        int32_t size_j,
        int32_t size_k,
        float const *a,
        float const *b,
        float *c,
        void *workspace) {
        matmul_improved_reduce::launch_matmul_improved_reduce(
            size_i,
            size_j,
            size_k,
            a,
            b,
            c,
            workspace);
    }
};

#endif

struct MatmulTensor {
    constexpr static char const *name = "matmul_tensor";

    static size_t get_workspace_size(int32_t size_i, int32_t size_j, int32_t size_k) {
        return matmul_tensor::get_workspace_size(size_i, size_j, size_k);
    }

    static void
    run(int32_t size_i,
        int32_t size_j,
        int32_t size_k,
        float const *a,
        float const *b,
        float *c,
        void *workspace) {
        matmul_tensor::launch_matmul_tensor(size_i, size_j, size_k, a, b, c, workspace);
    }
};

BenchmarkResults get_cublas_fma_results() {
    // Hard-coded data collected on A4000 GPU
    return BenchmarkResults{
        "cublas_fma",
        {
            {{3072, 3072, 3072}, 4.05},
            {{512, 3072, 3072}, 0.80},
            {{256, 3072, 3072}, 0.46},
            {{128, 3072, 3072}, 0.24},
            {{64, 3072, 3072}, 0.13},
            {{32, 3072, 3072}, 0.11},
            {{16, 3072, 3072}, 0.11},
        }};
}

std::vector<BenchmarkResults> run_all_impls(
    Phase phase,
    TestData const &data,
    std::vector<BenchmarkConfig> const &configs) {
    auto results = std::vector<BenchmarkResults>{};
#ifdef HAS_LAB_5_BASELINE_IMPL
    results.push_back(run_all_configs<MatmulImprovedReduce>(phase, data, configs));
#endif
    results.push_back(run_all_configs<MatmulTensor>(phase, data, configs));
    return results;
}

void write_json_results(
    std::string const &path,
    std::vector<BenchmarkResults> const &results) {
    auto file = std::ofstream(path);
    file << "{\n";
    for (int32_t i = 0; i < results.size(); ++i) {
        auto const &result = results.at(i);
        file << "  \"" << result.name << "\": [\n";
        int32_t j = 0;
        for (auto const &[config, elapsed_ms] : result.elapsed_ms) {
            auto [size_i, size_j, size_k] = config;
            double tflop = 2.0 * size_i * size_k * size_j * 1e-12;
            double tflop_per_sec = tflop / (elapsed_ms * 1e-3);
            file << "    {\n";
            file << "      \"size_i\": " << size_i << ",\n";
            file << "      \"size_j\": " << size_j << ",\n";
            file << "      \"size_k\": " << size_k << ",\n";
            file << "      \"elapsed_ms\": " << elapsed_ms << ",\n";
            file << "      \"tflop_per_sec\": " << tflop_per_sec << "\n";
            file << "    }";
            if (j + 1 < result.elapsed_ms.size()) {
                file << ",";
            }
            file << "\n";
            ++j;
        }
        file << "  ]";
        if (i + 1 < results.size()) {
            file << ",";
        }
        file << "\n";
    }
    file << "}\n";
}

void print_speedup(
    std::vector<BenchmarkConfig> const &configs,
    BenchmarkResults const &first,
    BenchmarkResults const &second) {
    printf("\nspeedups %s -> %s:\n\n", first.name, second.name);
    printf("  %-6s  %-6s  %-6s  %-7s\n", "size_i", "size_j", "size_k", "speedup");
    printf("  %-6s  %-6s  %-6s  %-7s\n", "------", "------", "------", "-------");
    for (auto const &config : configs) {
        auto size_i = config.size_i;
        auto size_j = config.size_j;
        auto size_k = config.size_k;
        printf("  %6d  %6d  %6d", size_i, size_j, size_k);
        auto it_first = first.elapsed_ms.find({size_i, size_j, size_k});
        auto it_second = second.elapsed_ms.find({size_i, size_j, size_k});
        if (it_first != first.elapsed_ms.end() && it_second != second.elapsed_ms.end()) {
            printf("  %6.02fx", it_first->second / it_second->second);
        } else {
            printf("  %7s", "-");
        }
        printf("\n");
    }
}

int main(int argc, char **argv) {
    std::string test_data_dir = ".";
    if (char *c_str_test_data_dir = std::getenv("MATMUL_TEST_DATA_DIR_2")) {
        test_data_dir = c_str_test_data_dir;
    }

    auto configs = std::vector<BenchmarkConfig>{
        {3072, 3072, 3072},
        {512, 3072, 3072},
        {256, 3072, 3072},
        {128, 3072, 3072},
        {64, 3072, 3072},
        {32, 3072, 3072},
        {16, 3072, 3072},
    };
    auto data = read_test_data(test_data_dir, configs);
    run_all_impls(Phase::WARMUP, data, configs);
    auto results = run_all_impls(Phase::BENCHMARK, data, configs);

    for (int32_t j = 1; j < results.size(); ++j) {
        for (int32_t i = j; i > 0;) {
            --i;
            print_speedup(configs, results.at(i), results.at(j));
        }
    }

    printf("\n-----------------------------------------------------------\n");
    printf("---- Comparison to non-tensor-core cuBLAS performance: ----\n");
    printf("-----------------------------------------------------------\n");

    print_speedup(configs, get_cublas_fma_results(), results.at(results.size() - 1));

    write_json_results("out/results.json", results);

    return 0;
}
