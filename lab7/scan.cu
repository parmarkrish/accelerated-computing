// Tested on RTX A4000
// nvcc -O3 -std=c++17 -gencode arch=compute_86,code=sm_86 -o scan scan.cu
#include <chrono>
#include <cstdint>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <vector>

////////////////////////////////////////////////////////////////////////////////
// Utility Functions

void cuda_check(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        std::cerr << "CUDA error at " << file << ":" << line << ": "
                  << hipGetErrorString(code) << std::endl;
        exit(1);
    }
}

#define CUDA_CHECK(x) \
    do { \
        cuda_check((x), __FILE__, __LINE__); \
    } while (0)

template <typename Op>
void print_array(
    size_t n,
    typename Op::Data const *x // allowed to be either a CPU or GPU pointer
);

////////////////////////////////////////////////////////////////////////////////
// CPU Reference Implementation (Already Written)

template <typename Op>
void scan_cpu(size_t n, typename Op::Data const *x, typename Op::Data *out) {
    using Data = typename Op::Data;
    Data accumulator = Op::identity();
    for (size_t i = 0; i < n; i++) {
        accumulator = Op::combine(accumulator, x[i]);
        out[i] = accumulator;
    }
}

/// <--- your code here --->

////////////////////////////////////////////////////////////////////////////////
// Optimized GPU Implementation

__device__ __forceinline__ int ceil_log2(uint32_t x) {
    return 32 - __clz(x - 1);
}

__device__ __forceinline__ int pow2(int i) {
    return 1 << i;
}

__device__ __forceinline__ void cp_async4(void *smem_ptr, const void *glob_ptr) {
    const int BYTES = 16;
    uint32_t smem = static_cast<uint32_t>(__cvta_generic_to_shared(smem_ptr));
    asm volatile(
        "cp.async.cg.shared.global [%0], [%1], %2;" ::"r"(smem),
        "l"(glob_ptr),
        "n"(BYTES));
}

__device__ __forceinline__ void async_memcpy_waitall() {
    asm volatile("cp.async.wait_all;\n" ::);
}

namespace scan_gpu {

constexpr int NUM_ELEMENTS_PER_THREAD = 12;

template <typename Op>
__device__ __forceinline__ void scan_per_thread(typename Op::Data* shmem, int start_index) {
    using Data = typename Op::Data;
    Data accum = Op::identity();

    // initial accumulation
    #pragma unroll
    for (int i = 0; i < NUM_ELEMENTS_PER_THREAD; i++) {
        accum = Op::combine(accum, shmem[start_index + i]);
        shmem[start_index + i] = accum;
    }
    __syncthreads();

    // continuing accumulations
    // int curr_idx = start_index - 1;
    for (int i = 0; i < ceil_log2(blockDim.x); i++) {
        int back_idx = NUM_ELEMENTS_PER_THREAD * (pow2(i) - 1) + 1;
        int curr_idx = start_index - back_idx;

        Data prev_accum = (curr_idx >= 0) ? shmem[curr_idx] : Op::identity();
        __syncthreads();

        #pragma unroll
        for (int j = 0; j < NUM_ELEMENTS_PER_THREAD; j++) {
            shmem[start_index + j] = Op::combine(prev_accum, shmem[start_index + j]);
        }
        __syncthreads();
    }
}

template <typename Op>
__device__ __forceinline__ void load_block_into_shmem(size_t n, typename Op::Data* shmem, typename Op::Data* gmem) {
    using Data = typename Op::Data;
    const int block_size = NUM_ELEMENTS_PER_THREAD * blockDim.x;
    constexpr int elems_per_vec_load = (16 / sizeof(Data));
    const int threadIdxi = elems_per_vec_load * threadIdx.x;
    const int i = blockIdx.x * block_size + threadIdxi;

    for (int shift_idx = 0; shift_idx < block_size; shift_idx += elems_per_vec_load * blockDim.x) {
        if (shift_idx + i < n) {
            cp_async4(shmem + shift_idx + threadIdxi, gmem + shift_idx + i);
        } else {
            #pragma unroll
            for (int i = 0; i < elems_per_vec_load; i++) {
                shmem[shift_idx + threadIdxi + i] = Op::identity();
            }
        }
    }
}


template <typename Op>
__global__ void scan_block(size_t n, typename Op::Data* src, typename Op::Data* dest, typename Op::Data* endpoints) {
    using Data = typename Op::Data;
    extern __shared__ __align__(16) char shmem_raw[];
    Data* shmem = reinterpret_cast<Data*>(shmem_raw);

    const int block_size = NUM_ELEMENTS_PER_THREAD * blockDim.x;
    // const int i = blockIdx.x * block_size + threadIdx.x;

    // load into shared memory
    // #pragma unroll
    // for (int shift_idx = 0; shift_idx < block_size; shift_idx += blockDim.x) {
    //     shmem[shift_idx + threadIdx.x] = (shift_idx + i < n) ? src[shift_idx + i] : Op::identity();
    // }
    load_block_into_shmem<Op>(n, shmem, src);
    async_memcpy_waitall();
    __syncthreads();

    // scan
    const int start_index = threadIdx.x * NUM_ELEMENTS_PER_THREAD;
    scan_per_thread<Op>(shmem, start_index);

    // store block
    #pragma unroll
    for (int shift_idx = 0; shift_idx < block_size; shift_idx += blockDim.x) {
        dest[blockIdx.x * block_size + shift_idx + threadIdx.x] = shmem[shift_idx + threadIdx.x];
    }

    if (gridDim.x == 1) return;  // if we have only one block, no need to perform a hierarchical scan
    
    // store endpoints
    if (threadIdx.x == blockDim.x - 1) {
        endpoints[blockIdx.x] = shmem[start_index + NUM_ELEMENTS_PER_THREAD - 1];
    }
}


template <typename Op>
__global__ void fixup(size_t n, typename Op::Data* workspace, typename Op::Data* endpoints) {
    using Data = typename Op::Data;

    if (blockIdx.x == 0) return;

    const int block_size = blockDim.x * NUM_ELEMENTS_PER_THREAD;
    const int j = blockIdx.x * block_size + threadIdx.x * NUM_ELEMENTS_PER_THREAD;
    const Data local_scan_val = endpoints[blockIdx.x - 1];

    #pragma unroll
    for (int i = 0; i < NUM_ELEMENTS_PER_THREAD; i++) {
        workspace[j+i] = Op::combine(local_scan_val, workspace[j+i]);
    }
}


// Returns desired size of scratch buffer in bytes.
template <typename Op> size_t get_workspace_size(size_t n) {
    using Data = typename Op::Data;
    constexpr int thread_per_block = 1024;
    const int block_size = NUM_ELEMENTS_PER_THREAD * thread_per_block;
    const int num_blocks = (n + (block_size - 1)) / block_size;
    return sizeof(Data) * ((num_blocks + 1) * block_size);
}

// 'launch_scan'
//
// Input:
//
//   'n': Number of elements in the input array 'x'.
//
//   'x': Input array in GPU memory. The 'launch_scan' function is allowed to
//   overwrite the contents of this buffer.
//
//   'workspace': Scratch buffer in GPU memory. The size of the scratch buffer
//   in bytes is determined by 'get_workspace_size<Op>(n)'.
//
// Output:
//
//   Returns a pointer to GPU memory which will contain the results of the scan
//   after all launched kernels have completed. Must be either a pointer to the
//   'x' buffer or to an offset within the 'workspace' buffer.
//
//   The contents of the output array should be "partial reductions" of the
//   input; each element 'i' of the output array should be given by:
//
//     output[i] = Op::combine(x[0], x[1], ..., x[i])
//
//   where 'Op::combine(...)' of more than two arguments is defined in terms of
//   repeatedly combining pairs of arguments. Note that 'Op::combine' is
//   guaranteed to be associative, but not necessarily commutative, so
//
//        Op::combine(a, b, c)              // conceptual notation; not real C++
//     == Op::combine(a, Op::combine(b, c)) // real C++
//     == Op::combine(Op::combine(a, b), c) // real C++
//
//  but we don't necessarily have
//
//    Op::combine(a, b) == Op::combine(b, a) // not true in general!
//
template <typename Op>
typename Op::Data *launch_scan(
    size_t n,
    typename Op::Data *x, // pointer to GPU memory
    void *workspace       // pointer to GPU memory
) {
    using Data = typename Op::Data;
    dim3 block_size_threads(1024);
    const int block_size = NUM_ELEMENTS_PER_THREAD * block_size_threads.x;
    dim3 num_blocks((n + block_size - 1) / block_size);

    uint32_t shmem_bytes = block_size * sizeof(Data);

    CUDA_CHECK(hipFuncSetAttribute(
        reinterpret_cast<const void*>(scan_block<Op>),
        hipFuncAttributeMaxDynamicSharedMemorySize, 
        shmem_bytes));

    Data* workspace_data = (Data*) workspace;
    Data* endpoints = workspace_data + num_blocks.x * block_size;

    scan_block<Op><<<num_blocks, block_size_threads, shmem_bytes>>>(n, x, workspace_data, endpoints);
    CUDA_CHECK(hipGetLastError());

    if (num_blocks.x > 1) {
        scan_block<Op><<<1, block_size_threads, shmem_bytes>>>((size_t) num_blocks.x, endpoints, endpoints, (Data*) NULL);
        CUDA_CHECK(hipGetLastError());

        fixup<Op><<<num_blocks, block_size_threads>>>(n, workspace_data, endpoints);
        CUDA_CHECK(hipGetLastError());
    }

    return workspace_data; // replace with an appropriate pointer
}

} // namespace scan_gpu

/// <--- /your code here --->

////////////////////////////////////////////////////////////////////////////////
///          YOU DO NOT NEED TO MODIFY THE CODE BELOW HERE.                  ///
////////////////////////////////////////////////////////////////////////////////

struct DebugRange {
    uint32_t lo;
    uint32_t hi;

    static constexpr uint32_t INVALID = 0xffffffff;

    static __host__ __device__ __forceinline__ DebugRange invalid() {
        return {INVALID, INVALID};
    }

    __host__ __device__ __forceinline__ bool operator==(const DebugRange &other) const {
        return lo == other.lo && hi == other.hi;
    }

    __host__ __device__ __forceinline__ bool operator!=(const DebugRange &other) const {
        return !(*this == other);
    }

    __host__ __device__ bool is_empty() const { return lo == hi; }

    __host__ __device__ bool is_valid() const { return lo != INVALID; }

    std::string to_string() const {
        if (lo == INVALID) {
            return "INVALID";
        } else {
            return std::to_string(lo) + ":" + std::to_string(hi);
        }
    }
};

struct DebugRangeConcatOp {
    using Data = DebugRange;

    static __host__ __device__ __forceinline__ Data identity() { return {0, 0}; }

    static __host__ __device__ __forceinline__ Data combine(Data a, Data b) {
        if (a.is_empty()) {
            return b;
        } else if (b.is_empty()) {
            return a;
        } else if (a.is_valid() && b.is_valid() && a.hi == b.lo) {
            return {a.lo, b.hi};
        } else {
            return Data::invalid();
        }
    }

    static std::string to_string(Data d) { return d.to_string(); }
};

struct SumOp {
    using Data = uint32_t;

    static __host__ __device__ __forceinline__ Data identity() { return 0; }

    static __host__ __device__ __forceinline__ Data combine(Data a, Data b) {
        return a + b;
    }

    static std::string to_string(Data d) { return std::to_string(d); }
};

constexpr size_t max_print_array_output = 1025;
static thread_local size_t total_print_array_output = 0;

template <typename Op> void print_array(size_t n, typename Op::Data const *x) {
    using Data = typename Op::Data;

    // copy 'x' from device to host if necessary
    hipPointerAttribute_t attr;
    CUDA_CHECK(hipPointerGetAttributes(&attr, x));
    auto x_host_buf = std::vector<Data>();
    Data const *x_host_ptr = nullptr;
    if (attr.type == hipMemoryTypeDevice) {
        x_host_buf.resize(n);
        x_host_ptr = x_host_buf.data();
        CUDA_CHECK(
            hipMemcpy(x_host_buf.data(), x, n * sizeof(Data), hipMemcpyDeviceToHost));
    } else {
        x_host_ptr = x;
    }

    if (total_print_array_output >= max_print_array_output) {
        return;
    }

    printf("[\n");
    for (size_t i = 0; i < n; i++) {
        auto s = Op::to_string(x_host_ptr[i]);
        printf("  [%zu] = %s,\n", i, s.c_str());
        total_print_array_output++;
        if (total_print_array_output > max_print_array_output) {
            printf("  ... (output truncated)\n");
            break;
        }
    }
    printf("]\n");

    if (total_print_array_output >= max_print_array_output) {
        printf("(Reached maximum limit on 'print_array' output; skipping further calls "
               "to 'print_array')\n");
    }

    total_print_array_output++;
}

template <typename Reset, typename F>
double benchmark_ms(double target_time_ms, Reset &&reset, F &&f) {
    double best_time_ms = std::numeric_limits<double>::infinity();
    double elapsed_ms = 0.0;
    while (elapsed_ms < target_time_ms) {
        reset();
        CUDA_CHECK(hipDeviceSynchronize());
        auto start = std::chrono::high_resolution_clock::now();
        f();
        CUDA_CHECK(hipDeviceSynchronize());
        auto end = std::chrono::high_resolution_clock::now();
        double this_ms = std::chrono::duration<double, std::milli>(end - start).count();
        elapsed_ms += this_ms;
        best_time_ms = std::min(best_time_ms, this_ms);
    }
    return best_time_ms;
}

struct Results {
    double time_ms;
    double bandwidth_gb_per_sec;
};

enum class Mode {
    TEST,
    BENCHMARK,
};

template <typename Op>
Results run_config(Mode mode, std::vector<typename Op::Data> const &x) {
    // Allocate buffers
    using Data = typename Op::Data;
    size_t n = x.size();
    size_t workspace_size = scan_gpu::get_workspace_size<Op>(n);
    Data *x_gpu;
    Data *workspace_gpu;
    CUDA_CHECK(hipMalloc(&x_gpu, n * sizeof(Data)));
    CUDA_CHECK(hipMalloc(&workspace_gpu, workspace_size));
    CUDA_CHECK(hipMemcpy(x_gpu, x.data(), n * sizeof(Data), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(workspace_gpu, 0, workspace_size));

    // Test correctness
    auto expected = std::vector<Data>(n);
    scan_cpu<Op>(n, x.data(), expected.data());
    auto out_gpu = scan_gpu::launch_scan<Op>(n, x_gpu, workspace_gpu);
    if (out_gpu == nullptr) {
        printf("'launch_scan' function not yet implemented (returned nullptr)\n");
        exit(1);
    }
    auto actual = std::vector<Data>(n);
    CUDA_CHECK(
        hipMemcpy(actual.data(), out_gpu, n * sizeof(Data), hipMemcpyDeviceToHost));
    for (size_t i = 0; i < n; ++i) {
        if (actual.at(i) != expected.at(i)) {
            auto actual_str = Op::to_string(actual.at(i));
            auto expected_str = Op::to_string(expected.at(i));
            printf(
                "Mismatch at position %zu: %s != %s\n",
                i,
                actual_str.c_str(),
                expected_str.c_str());
            if (n <= 128) {
                printf("Input:\n");
                print_array<Op>(n, x.data());
                printf("\nExpected:\n");
                print_array<Op>(n, expected.data());
                printf("\nActual:\n");
                print_array<Op>(n, actual.data());
            }
            exit(1);
        }
    }
    if (mode == Mode::TEST) {
        return {0.0, 0.0};
    }

    // Benchmark
    double target_time_ms = 200.0;
    double time_ms = benchmark_ms(
        target_time_ms,
        [&]() {
            CUDA_CHECK(
                hipMemcpy(x_gpu, x.data(), n * sizeof(Data), hipMemcpyHostToDevice));
            CUDA_CHECK(hipMemset(workspace_gpu, 0, workspace_size));
        },
        [&]() { scan_gpu::launch_scan<Op>(n, x_gpu, workspace_gpu); });
    double bytes_processed = n * sizeof(Data) * 2;
    double bandwidth_gb_per_sec = bytes_processed / time_ms / 1e6;

    // Cleanup
    CUDA_CHECK(hipFree(x_gpu));
    CUDA_CHECK(hipFree(workspace_gpu));

    return {time_ms, bandwidth_gb_per_sec};
}

std::vector<DebugRange> gen_debug_ranges(uint32_t n) {
    auto ranges = std::vector<DebugRange>();
    for (uint32_t i = 0; i < n; ++i) {
        ranges.push_back({i, i + 1});
    }
    return ranges;
}

template <typename Rng> std::vector<uint32_t> gen_random_data(Rng &rng, uint32_t n) {
    auto uniform = std::uniform_int_distribution<uint32_t>(0, 100);
    auto data = std::vector<uint32_t>();
    for (uint32_t i = 0; i < n; ++i) {
        data.push_back(uniform(rng));
    }
    return data;
}

template <typename Op, typename GenData>
void run_tests(std::vector<uint32_t> const &sizes, GenData &&gen_data) {
    for (auto size : sizes) {
        auto data = gen_data(size);
        printf("  Testing size %8u\n", size);
        run_config<Op>(Mode::TEST, data);
        printf("  OK\n\n");
    }
}

int main(int argc, char const *const *argv) {
    auto correctness_sizes = std::vector<uint32_t>{
        16,
        10,
        128,
        100,
        1024,
        1000,
        1 << 20,
        1'000'000,
        16 << 20,
        64 << 20,
    };

    auto rng = std::mt19937(0xCA7CAFE);

    printf("Correctness:\n\n");
    printf("Testing scan operation: debug range concatenation\n\n");
    run_tests<DebugRangeConcatOp>(correctness_sizes, gen_debug_ranges);
    printf("Testing scan operation: integer sum\n\n");
    run_tests<SumOp>(correctness_sizes, [&](uint32_t n) {
        return gen_random_data(rng, n);
    });

    printf("Performance:\n\n");

    size_t n = 64 << 20;
    auto data = gen_random_data(rng, n);

    printf("Benchmarking scan operation: integer sum, size %zu\n\n", n);

    // Warmup
    run_config<SumOp>(Mode::BENCHMARK, data);
    // Benchmark
    auto results = run_config<SumOp>(Mode::BENCHMARK, data);
    printf("  Time: %.2f ms\n", results.time_ms);
    printf("  Throughput: %.2f GB/s\n", results.bandwidth_gb_per_sec);

    return 0;
}